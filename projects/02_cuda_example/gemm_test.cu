#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <cutlass/util/host_tensor.h>
#include <cutlass/util/reference/host/tensor_compare.h>
#include <cutlass/util/reference/device/tensor_fill.h>
#include <cutlass/util/reference/host/gemm.h>
#include <iostream>
#include <vector>

using ElementType = float; // Data type for matrix elements
using LayoutType = cutlass::layout::RowMajor; // Row-major matrix layout
using Tensor = cutlass::HostTensor<ElementType, LayoutType>;

hipError_t device_gemm(int M, int N, int K,
                        Tensor& A,
                        Tensor& B,
                        Tensor& C,
                        ElementType alpha, ElementType beta){
        // Define CUTLASS GEMM kernel
        using Gemm = cutlass::gemm::device::Gemm<
                                                ElementType, LayoutType, // Matrix A: element type and layout
                                                ElementType, LayoutType, // Matrix B: element type and layout
                                                ElementType, LayoutType, // Matrix C: element type and layout
                                                ElementType>;            // Accumulator type

        // Configure GEMM operation
        Gemm gemm_op;
        cutlass::gemm::GemmCoord problem_size(M, N, K);

        // Define arguments
        typename Gemm::Arguments args(
                                    problem_size,                 // Problem size (M, N, K)
                                    {A.device_view(), K},                // Matrix A and leading dimension
                                    {B.device_view(), N},                // Matrix B and leading dimension
                                    {C.device_view(), N},                // Matrix C and leading dimension
                                    {C.device_view(), N},                // Matrix D (output matrix) and leading dimension
                                    {alpha, beta}                  // Scalars alpha and beta
        );
        cutlass::Status status = gemm_op.can_implement(args);
        if (status != cutlass::Status::kSuccess) {
            return hipErrorUnknown;
        }
        status = gemm_op(args);
        if (status != cutlass::Status::kSuccess) {
            return hipErrorUnknown;
        }
        
          return hipSuccess;
    }

int main() {
    
    // GEMM parameters
    int M = 128; // Rows of A and C
    int N = 128; // Columns of B and C
    int K = 128; // Columns of A and rows of B
    ElementType alpha = 1.0f;
    ElementType beta = 0.0f;
    
    Tensor A{cutlass::MatrixCoord(M, K)};
    Tensor B{cutlass::MatrixCoord(K, N)};
    Tensor C{cutlass::MatrixCoord(M, N)};
    Tensor C_ref{cutlass::MatrixCoord(M, N)};

    cutlass::reference::device::TensorFillRandomGaussian(A.device_view(), 10, 0, 1, 0);
    cutlass::reference::device::TensorFillRandomGaussian(B.device_view(), 10, 0, 1, 0);
    cutlass::reference::device::TensorFillRandomGaussian(C.device_view(), 10, 0, 1, 0);

    auto resutl = device_gemm(M, N, K, A, B, C, alpha, beta);

    A.sync_host();
    B.sync_host();
    C.sync_host();

    cutlass::reference::host::Gemm<
    ElementType,                           // ElementA
    LayoutType,              // LayoutA
    ElementType,                           // ElementB
    LayoutType,              // LayoutB
    ElementType,                           // ElementOutput
    LayoutType,              // LayoutOutput
    ElementType,
    ElementType
  > gemm_ref;

  gemm_ref(
    {M, N, K},                          // problem size (type: cutlass::gemm::GemmCoord)
    alpha,                              // alpha        (type: cutlass::half_t)
    A.host_ref(),                       // A            (type: TensorRef<half_t, ColumnMajor>)
    B.host_ref(),                       // B            (type: TensorRef<half_t, ColumnMajor>)
    beta,                               // beta         (type: cutlass::half_t)
    C_ref.host_ref()              // C            (type: TensorRef<half_t, ColumnMajor>)
  );

  if (!cutlass::reference::host::TensorEquals(
    C_ref.host_view(), 
    C.host_view())) {

    char const *filename = "errors_01_cutlass_utilities.csv";

    std::cerr << "Error - CUTLASS GEMM kernel differs from reference. Wrote computed and reference results to '" << filename << "'" << std::endl;
  } else {
    std::cout<< " The program finished successfully" <<std::endl;
  }

    return 0;
}
