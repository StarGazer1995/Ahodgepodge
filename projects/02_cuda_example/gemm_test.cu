#include <cutlass/cutlass.h>
#include <cutlass/gemm/device/gemm.h>
#include <iostream>
#include <vector>

// Check for CUDA errors
#define CHECK_CUDA(call)                                                              \
    {                                                                                 \
        hipError_t err = call;                                                       \
        if (err != hipSuccess) {                                                     \
            std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " : "     \
                      << hipGetErrorString(err) << std::endl;                        \
            exit(EXIT_FAILURE);                                                       \
        }                                                                             \
    }

int main() {
    using ElementType = float; // Data type for matrix elements
    using LayoutType = cutlass::layout::RowMajor; // Row-major matrix layout

    // GEMM parameters
    int M = 128; // Rows of A and C
    int N = 128; // Columns of B and C
    int K = 128; // Columns of A and rows of B

    // Host matrices
    std::vector<ElementType> host_A(M * K, 1.0f); // Initialize A with 1.0f
    std::vector<ElementType> host_B(K * N, 1.0f); // Initialize B with 1.0f
    std::vector<ElementType> host_C(M * N, 0.0f); // Initialize C with 0.0f

    // Device matrices
    ElementType *device_A, *device_B, *device_C;
    CHECK_CUDA(hipMalloc(&device_A, M * K * sizeof(ElementType)));
    CHECK_CUDA(hipMalloc(&device_B, K * N * sizeof(ElementType)));
    CHECK_CUDA(hipMalloc(&device_C, M * N * sizeof(ElementType)));

    // Copy data from host to device
    CHECK_CUDA(hipMemcpy(device_A, host_A.data(), M * K * sizeof(ElementType), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(device_B, host_B.data(), K * N * sizeof(ElementType), hipMemcpyHostToDevice));

    // Define CUTLASS GEMM kernel
    using Gemm = cutlass::gemm::device::Gemm<
        ElementType, LayoutType, // Matrix A: element type and layout
        ElementType, LayoutType, // Matrix B: element type and layout
        ElementType, LayoutType, // Matrix C: element type and layout
        ElementType>;            // Accumulator type

    // Configure GEMM operation
    Gemm gemm_op;
    cutlass::gemm::GemmCoord problem_size(M, N, K);

    // Define arguments
    typename Gemm::Arguments args(
        problem_size,                 // Problem size (M, N, K)
        {device_A, K},                // Matrix A and leading dimension
        {device_B, N},                // Matrix B and leading dimension
        {device_C, N},                // Matrix C and leading dimension
        {device_C, N},                // Matrix D (output matrix) and leading dimension
        {1.0f, 0.0f}                  // Scalars alpha and beta
    );

    // Check if the kernel is supported
    cutlass::Status status = gemm_op.can_implement(args);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "GEMM operation is not supported." << std::endl;
        return -1;
    }

    // Launch the GEMM kernel
    status = gemm_op(args);
    if (status != cutlass::Status::kSuccess) {
        std::cerr << "GEMM operation failed." << std::endl;
        return -1;
    }

    // Copy result back to host
    CHECK_CUDA(hipMemcpy(host_C.data(), device_C, M * N * sizeof(ElementType), hipMemcpyDeviceToHost));

    // Verify the results
    bool correct = true;
    for (int i = 0; i < M * N; ++i) {
        if (host_C[i] != K) { // Each element should be K (since A and B were filled with 1.0)
            correct = false;
            break;
        }
    }

    if (correct) {
        std::cout << "GEMM operation completed successfully!" << std::endl;
    } else {
        std::cerr << "GEMM operation failed verification!" << std::endl;
    }

    // Free device memory
    CHECK_CUDA(hipFree(device_A));
    CHECK_CUDA(hipFree(device_B));
    CHECK_CUDA(hipFree(device_C));

    return 0;
}
